
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
//#include <getopt.h>
#define HEADERSIZE 54   /* ヘッダのサイズ 54 = 14 + 40         */
#define PALLETSIZE 1024 /* パレットのサイズ                    */
#define MAXWIDTH 10000  /* 幅(pixel)の上限                     */
#define MAXHEIGHT 10000 /* 高さ(pixel) の上限                  */

/* x と y の交換のための マクロ関数 */
#define SWAP(x, y)      \
    {                   \
        typeof(x) temp; \
        temp = x;       \
        x = y;          \
        y = temp;       \
    }

#define BLOCK 5

unsigned char Bmp_headbuf[HEADERSIZE]; /* ヘッダを格納するための変数          */
unsigned char Bmp_Pallet[PALLETSIZE]; /* カラーパレットを格納                */

char Bmp_type[2];       /* ファイルタイプ "BM"                 */
unsigned long Bmp_size; /* bmpファイルのサイズ (バイト)        */
unsigned int Bmp_info_header_size; /* 情報ヘッダのサイズ = 40             */
unsigned int Bmp_header_size;      /* ヘッダサイズ = 54*/
long Bmp_height;           /* 高さ (ピクセル)                     */
long Bmp_width;            /* 幅   (ピクセル)                     */
unsigned short Bmp_planes; /* プレーン数 常に 1                   */
unsigned short Bmp_color;  /* 色 (ビット)     24                  */
long Bmp_comp;             /* 圧縮方法         0                  */
long Bmp_image_size; /* 画像部分のファイルサイズ (バイト)   */
long Bmp_xppm;       /* 水平解像度 (ppm)                    */
long Bmp_yppm;       /* 垂直解像度 (ppm)                    */

typedef struct { /* 1ピクセルあたりの赤緑青の各輝度     */
    unsigned char r;
    unsigned char g;
    unsigned char b;
} color;

typedef struct {
    long height;
    long width;
    color data[MAXHEIGHT][MAXWIDTH];
} img;
/*
   関数名: ReadBmp
   引数  : char *filename, img *imgp
   返り値: void
   動作  : bmp形式のファイル filename を開いて, その画像データを
           2次元配列 imgp->data に格納する. 同時に, ヘッダから読み込まれた
           画像の幅と高さをグローバル変数 Bmp_width とBmp_height にセットする.
*/
void ReadBmp(const char *filename, img *imgp) {
    int i, j;
    int Real_width;
    FILE *Bmp_Fp =
        fopen(filename, "rb"); /* バイナリモード読み込み用にオープン  */
    unsigned char *Bmp_Data; /* 画像データを1行分格納               */

    if (Bmp_Fp == NULL) {
        fprintf(stderr, "Error: file %s couldn\'t open for read!.\n", filename);
        exit(1);
    }

    /* ヘッダ読み込み */
    fread(Bmp_headbuf, sizeof(unsigned char), HEADERSIZE, Bmp_Fp);

    memcpy(&Bmp_type, Bmp_headbuf, sizeof(Bmp_type));
    if (strncmp(Bmp_type, "BM", 2) != 0) {
        fprintf(stderr, "Error: %s is not a bmp file.\n", filename);
        exit(1);
    }

    memcpy(&imgp->width, Bmp_headbuf + 18, sizeof(Bmp_width));
    memcpy(&imgp->height, Bmp_headbuf + 22, sizeof(Bmp_height));
    memcpy(&Bmp_color, Bmp_headbuf + 28, sizeof(Bmp_color));
    if (Bmp_color != 24) {
        fprintf(stderr,
                "Error: Bmp_color = %d is not implemented in this program.\n",
                Bmp_color);
        exit(1);
    }

    if (imgp->width > MAXWIDTH) {
        fprintf(stderr, "Error: Bmp_width = %ld > %d = MAXWIDTH!\n", Bmp_width,
                MAXWIDTH);
        exit(1);
    }

    if (imgp->height > MAXHEIGHT) {
        fprintf(stderr, "Error: Bmp_height = %ld > %d = MAXHEIGHT!\n",
                Bmp_height, MAXHEIGHT);
        exit(1);
    }

    Real_width = imgp->width * 3 +
                 imgp->width % 4; /* 4byte 境界にあわせるために実際の幅の計算 */

    /* 配列領域の動的確保. 失敗した場合はエラーメッセージを出力して終了 */
    if ((Bmp_Data = (unsigned char *)calloc(Real_width,
                                            sizeof(unsigned char))) == NULL) {
        fprintf(stderr, "Error: Memory allocation failed for Bmp_Data!\n");
        exit(1);
    }

    /* 画像データ読み込み */
    for (i = 0; i < imgp->height; i++) {
        fread(Bmp_Data, 1, Real_width, Bmp_Fp);
        for (j = 0; j < imgp->width; j++) {
            imgp->data[imgp->height - i - 1][j].b = Bmp_Data[j * 3];
            imgp->data[imgp->height - i - 1][j].g = Bmp_Data[j * 3 + 1];
            imgp->data[imgp->height - i - 1][j].r = Bmp_Data[j * 3 + 2];
        }
    }

    /* 動的に確保した配列領域の解放 */
    free(Bmp_Data);

    /* ファイルクローズ */
    fclose(Bmp_Fp);
}

/*
   関数名: WriteBmp
   引数  : char *filename, img *tp
   返り値: void
   動作  : 2次元配列 tp->data の内容を画像データとして, 24ビット
           bmp形式のファイル filename に書き出す.
*/
void WriteBmp(const char *filename, img *tp) {
    int i, j;
    int Real_width;
    FILE *Out_Fp = fopen(filename, "wb"); /* ファイルオープン */
    unsigned char *Bmp_Data; /* 画像データを1行分格納               */

    if (Out_Fp == NULL) {
        fprintf(stderr, "Error: file %s couldn\'t open for write!\n", filename);
        exit(1);
    }

    Bmp_color = 24;
    Bmp_header_size = HEADERSIZE;
    Bmp_info_header_size = 40;
    Bmp_planes = 1;

    Real_width = tp->width * 3 +
                 tp->width % 4; /* 4byte 境界にあわせるために実際の幅の計算 */

    /* 配列領域の動的確保. 失敗した場合はエラーメッセージを出力して終了 */
    if ((Bmp_Data = (unsigned char *)calloc(Real_width,
                                            sizeof(unsigned char))) == NULL) {
        fprintf(stderr, "Error: Memory allocation failed for Bmp_Data!\n");
        exit(1);
    }

    /* ヘッダ情報の準備 */
    Bmp_xppm = Bmp_yppm = 0;
    Bmp_image_size = tp->height * Real_width;
    Bmp_size = Bmp_image_size + HEADERSIZE;
    Bmp_headbuf[0] = 'B';
    Bmp_headbuf[1] = 'M';
    memcpy(Bmp_headbuf + 2, &Bmp_size, sizeof(Bmp_size));
    Bmp_headbuf[6] = Bmp_headbuf[7] = Bmp_headbuf[8] = Bmp_headbuf[9] = 0;
    memcpy(Bmp_headbuf + 10, &Bmp_header_size, sizeof(Bmp_header_size));
    Bmp_headbuf[11] = Bmp_headbuf[12] = Bmp_headbuf[13] = 0;
    memcpy(Bmp_headbuf + 14, &Bmp_info_header_size,
           sizeof(Bmp_info_header_size));
    Bmp_headbuf[15] = Bmp_headbuf[16] = Bmp_headbuf[17] = 0;
    memcpy(Bmp_headbuf + 18, &tp->width, sizeof(Bmp_width));
    memcpy(Bmp_headbuf + 22, &tp->height, sizeof(Bmp_height));
    memcpy(Bmp_headbuf + 26, &Bmp_planes, sizeof(Bmp_planes));
    memcpy(Bmp_headbuf + 28, &Bmp_color, sizeof(Bmp_color));
    memcpy(Bmp_headbuf + 34, &Bmp_image_size, sizeof(Bmp_image_size));
    memcpy(Bmp_headbuf + 38, &Bmp_xppm, sizeof(Bmp_xppm));
    memcpy(Bmp_headbuf + 42, &Bmp_yppm, sizeof(Bmp_yppm));
    Bmp_headbuf[46] = Bmp_headbuf[47] = Bmp_headbuf[48] = Bmp_headbuf[49] = 0;
    Bmp_headbuf[50] = Bmp_headbuf[51] = Bmp_headbuf[52] = Bmp_headbuf[53] = 0;

    /* ヘッダ情報書き出し */
    fwrite(Bmp_headbuf, sizeof(unsigned char), HEADERSIZE, Out_Fp);

    /* 画像データ書き出し */
    for (i = 0; i < tp->height; i++) {
        for (j = 0; j < tp->width; j++) {
            Bmp_Data[j * 3] = tp->data[tp->height - i - 1][j].b;
            Bmp_Data[j * 3 + 1] = tp->data[tp->height - i - 1][j].g;
            Bmp_Data[j * 3 + 2] = tp->data[tp->height - i - 1][j].r;
        }
        for (j = tp->width * 3; j < Real_width; j++) {
            Bmp_Data[j] = 0;
        }
        fwrite(Bmp_Data, sizeof(unsigned char), Real_width, Out_Fp);
    }

    /* 動的に確保した配列領域の解放 */
    free(Bmp_Data);

    /* ファイルクローズ */
    fclose(Out_Fp);
}

#define PI 3.1415

__global__ void GPU_process(img *picture_p, img *output_p, double rad,
                            int add_width, int add_height, int height,
                            int width) {
    int raw_y = blockIdx.x * blockDim.x + threadIdx.x;
    int raw_x = blockIdx.y * blockDim.y + threadIdx.y;
    if (raw_x < 0 || raw_y < 0 || raw_x >= width || raw_y >= height) {
        return;
    }
    for (double deg = 0; deg < 360; deg++) {
        double Rad = deg * PI / 180.0;
        int x = raw_x - picture_p->width / 2;
        int y = raw_y - picture_p->height / 2;
        int new_x = -x * cos(Rad) + y * sin(Rad);
        int new_y = y * cos(Rad) + x * sin(Rad);
        new_x += add_width;
        new_y += add_height;
        output_p->data[new_y][new_x].r = picture_p->data[raw_y][raw_x].r;
        output_p->data[new_y][new_x].g = picture_p->data[raw_y][raw_x].g;
        output_p->data[new_y][new_x].b = picture_p->data[raw_y][raw_x].b;
    }
}

int main(int argc, char *argv[]) {
    double deg, rad;
    int add_height, add_width;
    img *picture_p;
    picture_p = (img *)malloc(sizeof(img));
    if (argc != 2) {
        printf("init_CUDA.exe file_name\n");
        exit(0);
    }
    ReadBmp(argv[1], picture_p);

    // 4点を見る
    //(0,0) (0,width) (height,0) (height,width)
    deg = 45.0;
    rad = deg * PI / 180.0;
    int min_height = 1e9;
    int max_height = -1e9;
    int min_width = 1e9;
    int max_width = -1e9;
    int four_check[4][2] = {{0, 0},
                            {0, picture_p->width},
                            {picture_p->height, 0},
                            {picture_p->height, picture_p->width}};
    for (int i = 0; i < 4; i++) {
        int height = four_check[i][0];
        height -= picture_p->height / 2;
        int width = four_check[i][1];
        width -= picture_p->width / 2;
        int new_height = height * cos(rad) + width * sin(rad);
        int new_width = -height * sin(rad) + width * cos(rad);
        if (new_height > max_height) {
            max_height = new_height;
        }
        if (new_height < min_height) {
            min_height = new_height;
        }
        if (new_width > max_width) {
            max_width = new_width;
        }
        if (new_width < min_width) {
            min_width = new_width;
        }
    }
    max_height -= min_height;
    max_width -= min_width;
    add_height = -min_height;
    add_width = -min_width;

    img *output_p;
    output_p = (img *)malloc(sizeof(img));
    output_p->height = max_height;
    output_p->width = max_width;

    img * cuda_output;
    hipMalloc(&cuda_output,sizeof(img));
    img * cuda_input;
    hipMalloc(&cuda_input,sizeof(img));
    hipMemcpy(cuda_input,picture_p,sizeof(img),hipMemcpyHostToDevice);
    dim3 grid((picture_p->height+BLOCK)/BLOCK, (picture_p->width+BLOCK)/BLOCK,1); 
    dim3 threads(BLOCK,BLOCK,1);
    GPU_process<<<grid,threads>>>(cuda_input,cuda_output,rad,add_width,add_height,picture_p->height,picture_p->width);

    printf("end\n");
    hipMemcpy(output_p,cuda_output,sizeof(img),hipMemcpyDeviceToHost);
    hipFree(cuda_output);
    output_p->height=max_height;
    output_p->width=max_width;
    /*
    for (double Deg = 0; Deg < 360; Deg++) {
        double Rad = Deg * PI / 180.0;
        for (int raw_y = 0; raw_y < picture_p->height; raw_y++) {
            for (int raw_x = 0; raw_x < picture_p->width; raw_x++) {
                int x = raw_x - picture_p->width / 2;
                int y = raw_y - picture_p->height / 2;
                int new_x = -x * cos(Rad) + y * sin(Rad);
                int new_y = y * cos(Rad) + x * sin(Rad);
                new_x += add_width;
                new_y += add_height;
                if (new_x < 0) new_x = 0;
                if (new_y < 0) new_y = 0;
                output_p->data[new_y][new_x].r =
                    picture_p->data[raw_y][raw_x].r;
                output_p->data[new_y][new_x].g =
                    picture_p->data[raw_y][raw_x].g;
                output_p->data[new_y][new_x].b =
                    picture_p->data[raw_y][raw_x].b;
            }
        }
    }*/
    WriteBmp("CUDA_output/gpu_output4.bmp", output_p);
    free(output_p);
    return 0;
}
